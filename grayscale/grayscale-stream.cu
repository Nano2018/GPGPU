#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>
#include <vector>
using namespace std;
__global__ void grayscale(unsigned char *rgb, unsigned char *out, std::size_t rows, std::size_t cols){
	auto idx = blockIdx.x * blockDim.x + threadIdx.x;
	auto idy = blockIdx.y * blockDim.y + threadIdx.y;
	if(idx < rows && idy < cols){
		out[idx * cols + idy] = (
			 307 * rgb[ 3 * ( idx * cols + idy ) ]
		       + 604 * rgb[ 3 * ( idx * cols + idy ) + 1 ]
		       + 113 * rgb[  3 * ( idx * cols + idy ) + 2 ]
		       ) / 1024;
	}
}

int main()
{
  cv::Mat m_in = cv::imread("4v9mo.jpg", cv::IMREAD_UNCHANGED );
  auto rgb = m_in.data;
  auto rows = m_in.rows;
  auto cols = m_in.cols;
  auto bpp = 3;
  auto size_img = cols*rows*bpp;
  auto size_matrix = cols*rows;
  hipHostRegister( rgb,size_img, hipHostRegisterDefault);
  std::vector< unsigned char > out( size_matrix );
  cv::Mat m_out(rows,cols, CV_8UC1, out.data());
  unsigned char* out_d;
  unsigned char* data_d;
  hipError_t err = hipMalloc( &out_d, size_matrix );
  if( err != hipSuccess ) {
    cerr << "Error hipMalloc: " << hipGetErrorString(err) << endl;
    exit(EXIT_FAILURE);
  }
  err = hipMalloc(&data_d,size_img+(2*size_matrix));
  if( err != hipSuccess ) {
    cerr << "Error hipMalloc: " << hipGetErrorString(err) << endl;
    exit(EXIT_FAILURE);
  }
  hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );
  hipEventRecord( start );
  //streams:
  hipStream_t streams[2];
  for( std::size_t i = 0 ; i < 2 ; ++i ){
    hipStreamCreate(&streams[ i ] );
  }

  auto size = bpp*cols;
  auto offset = 0;
  for( size_t i = 0 ; i < 2 ; ++i ){
    if(i == 1){
      offset = bpp*cols;
    }
    hipMemcpyAsync(data_d + i*size_img/2+offset , rgb + i*size_img/2-offset , size_img/2+size, hipMemcpyHostToDevice, streams[i] );
    /*hipDeviceSynchronize();
    err = hipGetLastError();
    if( err != hipSuccess ) {
      cerr << "Error hipMemcpyAsync: " << hipGetErrorString(err) << endl;
      exit(EXIT_FAILURE);
    }*/
   }

  //lancement du kernel avec les streams.
  dim3 t( 32, 32 );
  dim3 b( ( rows - 1) / (t.x) + 1 , ( cols - 1 ) / (t.y) + 1 );
  for(size_t i=0; i<2; i++){
     grayscale<<< b, t, 0,streams[i] >>>( data_d + i*size_img/2, out_d + i*size_matrix/2, rows/2+1, cols );
     /*hipDeviceSynchronize();
     err = hipGetLastError();
     if(err != hipSuccess){
        std::cout << "Error kernel : " << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
     }*/
  }

  for( size_t i = 0 ; i < 2 ; ++i ){
    hipMemcpyAsync(out.data() + i*size_matrix/2, out_d + i*size_matrix/2 ,size_matrix/2, hipMemcpyDeviceToHost, streams[i] );
    /*hipDeviceSynchronize();
    err = hipGetLastError();
    if( err != hipSuccess ) {
        cerr << "Error cudaMemcpyAsyncDeviceToHost: " << hipGetErrorString(err)<< endl;
        exit(EXIT_FAILURE);
    }*/
   }
  hipEventRecord( stop );
  hipEventSynchronize( stop );
  float duration = 0.0f;
  hipEventElapsedTime( &duration, start, stop );
  cout << "time: " << duration << "ms\n";
  cv::imwrite( "out.jpg", m_out );
  hipFree(data_d);
  hipFree(out_d);
  return 0;
}
