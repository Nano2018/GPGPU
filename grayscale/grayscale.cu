#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>
#include <vector>
using namespace std;
__global__ void grayscale(unsigned char *rgb, unsigned char *out, std::size_t rows, std::size_t cols){
	auto idx = blockIdx.x * blockDim.x + threadIdx.x;
	auto idy = blockIdx.y * blockDim.y + threadIdx.y;
	if(idx < rows && idy < cols){
		out[idx * cols + idy] = (
			 307 * rgb[ 3 * ( idx * cols + idy ) ]
		       + 604 * rgb[ 3 * ( idx * cols + idy ) + 1 ]
		       + 113 * rgb[  3 * ( idx * cols + idy ) + 2 ]
		       ) / 1024;
	}
}

int main()
{
  cv::Mat m_in = cv::imread("4v9mo.jpg", cv::IMREAD_UNCHANGED );
  auto rgb = m_in.data;
  auto rows = m_in.rows;
  auto cols = m_in.cols;
  std::vector< unsigned char > g( rows * cols );
  cv::Mat m_out(rows,cols, CV_8UC1, g.data());
  unsigned char *rgb_d = nullptr;
  unsigned char *m_d = nullptr;
  hipMalloc(&m_d, rows * cols);
  hipMalloc(&rgb_d, 3 * rows * cols );
  hipMemcpy(rgb_d,rgb,rows * cols * 3,hipMemcpyHostToDevice);

  hipEvent_t start, stop;
  hipEventCreate( &start );
  hipEventCreate( &stop );
  hipEventRecord( start );
  dim3 block(32,32);
  dim3 grid( ( rows - 1) / block.x + 1 , ( cols - 1 ) /block.y + 1 );
  grayscale<<<grid,block>>>(rgb_d,m_d,rows,cols);
  //gestion des erreurs.
  /*hipDeviceSynchronize();
  hipError_t err = hipGetLastError();     
  if(err != hipSuccess){
	cerr << hipGetErrorString(err) << endl;
	exit(EXIT_FAILURE);
  }*/
	hipMemcpy(g.data(),m_d,rows * cols,hipMemcpyDeviceToHost);
  hipEventRecord( stop );
  hipEventSynchronize( stop );
  float duration = 0.0f;
  hipEventElapsedTime( &duration, start, stop );
  cout << "time: " << duration << "ms\n";
	//hipMemcpy(g.data(),m_d,rows * cols,hipMemcpyDeviceToHost);
  cv::imwrite( "out.jpg", m_out );
  hipFree(rgb_d);
  hipFree(m_d);
  return 0;
}
